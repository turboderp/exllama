#define _cuda_buffers_cu
#include "cuda_buffers.cuh"

CudaBuffers* g_buffers[CUDA_MAX_DEVICES] = {NULL};
// __constant__ half2 q4_table[16][256];
// half2 q4_table_host[16][256];
// bool q4_table_init = false;

CudaBuffers::CudaBuffers
(
    int _device,
    half* _temp_state,
    int _temp_state_size,
    half* _temp_mlp,
    float* _temp_zeros_float,
    half* _temp_dq,
    int _max_zeros_float
) :
    device(_device),
    temp_state(_temp_state),
    temp_state_size(_temp_state_size),
    temp_mlp(_temp_mlp),
    temp_zeros_float(_temp_zeros_float),
    temp_dq(_temp_dq),
    max_zeros_float(_max_zeros_float),
    current_zeros_float(0)
{
    hipSetDevice(_device);

    hipStreamCreate(&alt_stream_1);
    hipStreamCreate(&alt_stream_2);
    hipStreamCreate(&alt_stream_3);
    hipEventCreate(&alt_stream_1_done);
    hipEventCreate(&alt_stream_2_done);
    hipEventCreate(&alt_stream_3_done);
}

CudaBuffers::~CudaBuffers()
{
    hipStreamDestroy(alt_stream_1);
    hipStreamDestroy(alt_stream_2);
    hipStreamDestroy(alt_stream_3);
    hipEventDestroy(alt_stream_1_done);
    hipEventDestroy(alt_stream_2_done);
    hipEventDestroy(alt_stream_3_done);
}

float* CudaBuffers::get_zeros_float(const int num_zeros)
{
    if (current_zeros_float + num_zeros >= max_zeros_float)
    {
        current_zeros_float = 0;
        hipMemsetAsync(temp_zeros_float, 0, max_zeros_float * sizeof(float));
    }

    float* zeros = temp_zeros_float + current_zeros_float;
    current_zeros_float += num_zeros;
    return zeros;
}

CudaBuffers* get_buffers(const int device_index)
{
    return g_buffers[device_index];
}

void prepare_buffers_cuda
(
    int _device,
    half* _temp_state,
    int _temp_state_size,
    half* _temp_mlp,
    float* _temp_zeros_float,
    half* _temp_dq,
    int _max_zeros_float
)
{
    CudaBuffers* buffers = new CudaBuffers
    (
        _device,
        _temp_state,
        _temp_state_size,
        _temp_mlp,
        _temp_zeros_float,
        _temp_dq,
        _max_zeros_float
    );

    g_buffers[_device] = buffers;
}

void cleanup_buffers_cuda()
{
    for (int i = 0; i < CUDA_MAX_DEVICES; i++)
    {
        if (!g_buffers[i]) continue;
        delete g_buffers[i];
        g_buffers[i] = NULL;
    }
}

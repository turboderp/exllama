#include "hip/hip_runtime.h"
#include "half_matmul.cuh"
#include "../util.cuh"
#include "../matrix.cuh"
#include "../cuda_compat.cuh"
#if defined(USE_ROCM)
#include "../hip_compat.cuh"
#endif

// Block size

const int THREADS_X = 32;     // Block size and thread count along columns in w and out
const int THREADS_Y = 8;      // Block size and thread count along rows in x and out
const int BLOCKSIZE = 256;

__global__ void half_matmul_kernel
(
    const half* __restrict__ x,
    const half* __restrict__ w,
    half* __restrict__ out,
    const int height,
    const int dim,
    const int width
)
{
    const int column = (blockIdx.x * THREADS_X + threadIdx.x) * 2;
    const int row = blockIdx.y * THREADS_Y + threadIdx.y;
    const int k0 = blockIdx.z * BLOCKSIZE;

    if (row >= height) return;
    if (column >= width) return;

    MatrixView_half x_(x, height, dim);
    MatrixView_half w_(w, dim, width);
    MatrixView_half_rw out_(out, height, width);

    half2* x_ptr = (half2*) x_.item_ptr(row, k0);
    half2* w_ptr = (half2*) w_.item_ptr(k0, column);
    half2 acc = {};

    #pragma unroll
    for (int k = k0; k < k0 + BLOCKSIZE / 2; k++)
    {
        half2 x_item = *x_ptr++;
        half2 x_item_0 = __low2half2(x_item);
        half2 x_item_1 = __high2half2(x_item);
        half2 w_item_0 = *w_ptr; w_ptr += w_.width / 2;
        half2 w_item_1 = *w_ptr; w_ptr += w_.width / 2;
        acc = __hfma2(x_item_0, w_item_0, acc);
        acc = __hfma2(x_item_1, w_item_1, acc);
    }

    // out_.set(row, column, acc);
    atomicAdd((half2*)out_.item_ptr(row, column), acc);
}

void half_matmul_cuda
(
    const half* x,
    const half* w,
    half* out,
    const int height,
    const int dim,
    const int width,
    hipStream_t alt_stream
)
{
    dim3 threads(THREADS_X, THREADS_Y, 1);

    dim3 blocks
    (
        (width + THREADS_X - 1) / THREADS_X / 2,
        (height + THREADS_Y - 1) / THREADS_Y,
        (dim + BLOCKSIZE - 1) / BLOCKSIZE
    );

    half_matmul_kernel<<<blocks, threads, 0, alt_stream>>>(x, w, out, height, dim, width);
}

// cuBLAS can't be beat for large matrices, probably

const int MAX_DIM_SMALL = 8192;

void half_matmul_cublas_cuda
(
    ExLlamaTuning* tuningParams,
    const half* x,
    const half* w,
    half* out,
    const int height,
    const int dim,
    const int width,
    hipblasHandle_t handle,
    bool no_zero,
    hipStream_t alt_stream
)
{
    // Fall back on a naive kernel for small matmuls to avoid cuBLAS overhead

    if (height < 4 && dim <= MAX_DIM_SMALL)
    {
        half_matmul_small_cuda(tuningParams, x, w, out, height, dim, width, no_zero, alt_stream);
        return;
    }

    // printf("cuBLAS: (%i, %i) @ (%i, %i) -> (%i, %i)\n", height, dim, dim, width, height, width);

    // Use cuBLAS

    const half alpha = __float2half(1.0f);
    const half beta = no_zero ? __float2half(1.0f) : __float2half(0.0f);

    hipStream_t default_stream;
    if (alt_stream)
    {
        hipblasGetStream(handle, &default_stream);
        hipblasSetStream(handle, alt_stream);
    }

    hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, width, height, dim, &alpha, w, width, x, dim, &beta, out, width);

    if (alt_stream)
    {
        hipblasSetStream(handle, default_stream);
    }
}

// Alternative to cuBLAS for tall or wide matrices

const int S_THREADS_X = 8;                                      // width
const int S_THREADS_Z = 1;                                      // height
const int S_BLOCKSIZE = MAX_DIM_SMALL / 1024 * S_THREADS_X;     // dim

template<bool use_half2, bool odd_rank>
__global__ void half_matmul_small_kernel
(
    const half* __restrict__ x,
    const half* __restrict__ w,
    half* __restrict__ out,
    const int height,
    const int dim,
    const int width,
    bool no_zero
)
{
    int column = blockIdx.x * S_THREADS_X + threadIdx.x;
    int row = blockIdx.z * S_THREADS_Z + threadIdx.z;
    int k = threadIdx.y * S_BLOCKSIZE;

    if (row >= height) return;
    if (column >= width) return;
    // if (k >= dim) return;
    // printf("%i, %i, %i\n", row, column, k);

    MatrixView_half x_(x, height, dim);
    MatrixView_half w_(w, dim, width);
    MatrixView_half_rw out_(out, height, width);

    int k_end = k + S_BLOCKSIZE;
    if (k_end > dim) k_end = dim;

    const half* x_ptr = x_.item_ptr(row, k);
    const half* x_ptr_end = x_.item_ptr(row, k_end);
    const half* w_ptr = w_.item_ptr(k, column);
    half* out_ptr = out_.item_ptr(row, column);

    if constexpr (use_half2 && !odd_rank)
    {
        half2* x_ptr2 = (half2*) x_ptr;
        half2* x_ptr2_end = (half2*) x_ptr_end;

        half2 r = {};

        while(x_ptr2 < x_ptr2_end)
        {
            half2 x_01 = *x_ptr2++;
            half2 x_23 = *x_ptr2++;
            half w_0 = *w_ptr; w_ptr += width;
            half w_1 = *w_ptr; w_ptr += width;
            half w_2 = *w_ptr; w_ptr += width;
            half w_3 = *w_ptr; w_ptr += width;
            half2 w_01 = __halves2half2(w_0, w_1);
            half2 w_23 = __halves2half2(w_2, w_3);
            r = __hfma2(x_01, w_01, r);
            r = __hfma2(x_23, w_23, r);
        }

        half rh = __hadd(__low2half(r), __high2half(r));

        __shared__ half accum[MAX_DIM_SMALL / S_BLOCKSIZE][S_THREADS_X];
        accum[threadIdx.y][threadIdx.x] = rh;
        __syncthreads();

        if (threadIdx.y == 0)
        {
            half acc = rh;
            for (int i = 1; i < blockDim.y; ++i) acc = __hadd(accum[i][threadIdx.x], acc);
            if (no_zero) acc = __hadd(acc, *out_ptr);
            *out_ptr = acc;
        }
    }
    else
    {
        half r = {};

        while(x_ptr < x_ptr_end)
        {
            if constexpr (odd_rank)
            {
                half x_item = *x_ptr++;
                half w_item = *w_ptr; w_ptr += width;
                r = __hfma(x_item, w_item, r);
            }
            else
            {
                #pragma unroll
                for (int i = 0; i < 4; ++i)
                {
                    half x_item = *x_ptr++;
                    half w_item = *w_ptr; w_ptr += width;
                    r = __hfma(x_item, w_item, r);
                }
            }
        }

        __shared__ half accum[MAX_DIM_SMALL / S_BLOCKSIZE][S_THREADS_X];
        accum[threadIdx.y][threadIdx.x] = r;
        __syncthreads();

        if (threadIdx.y == 0)
        {
            half acc = accum[0][threadIdx.x];
            for (int i = 1; i < blockDim.y; ++i) acc = __hadd(accum[i][threadIdx.x], acc);
            if (no_zero) acc = __hadd(acc, *out_ptr);
            *out_ptr = acc;
        }
    }
}

void half_matmul_small_cuda
(
    ExLlamaTuning* tuningParams,
    const half* x,
    const half* w,
    half* out,
    const int height,
    const int dim,
    const int width,
    bool no_zero,
    hipStream_t alt_stream
)
{
    bool use_half2 = !tuningParams->matmul_no_half2;

    //printf("kernel: (%i, %i) @ (%i, %i) -> (%i, %i)\n", height, dim, dim, width, height, width);

    dim3 threads
    (
        S_THREADS_X,
        (dim + S_BLOCKSIZE - 1) / S_BLOCKSIZE,
        1
    );

    dim3 blocks
    (
        (width + S_THREADS_X - 1) / S_THREADS_X,
        1,
        height
    );

    //printf("t... %i %i %i\n", threads.x, threads.y, threads.z);
    //printf("b... %i %i %i\n", blocks.x, blocks.y, blocks.z);
    //if (!no_zero) hipMemsetAsync(out, 0, height * width * sizeof(half));

    if (dim & 0x03)
    {
        half_matmul_small_kernel<false, true> <<<blocks, threads, 0, alt_stream>>>(x, w, out, height, dim, width, no_zero);
    }
    else
    {
        if (use_half2) half_matmul_small_kernel<true,  false> <<<blocks, threads, 0, alt_stream>>>(x, w, out, height, dim, width, no_zero);
        else           half_matmul_small_kernel<false, false> <<<blocks, threads, 0, alt_stream>>>(x, w, out, height, dim, width, no_zero);
    }
}


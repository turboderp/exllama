#include "hip/hip_runtime.h"
#include "q4_matmul.cuh"
#include "column_remap.cuh"
#include "../util.cuh"
#include "../matrix.cuh"
#include "../cuda_compat.cuh"
#include "../cuda_buffers.cuh"
#if defined(USE_ROCM)
#include "../hip_compat.cuh"
#endif

const int THREADS_X = 32;       // Block size and thread count along columns in w and out
const int THREADS_Y = 1;        // Block size and thread count along rows in x and out

template<bool use_half2, bool use_groupsize, bool use_x_map>
__global__ void q4_matmul_kernel
(
    const half* x,
    const uint32_t* w,
    half* out,  // (y)
    const half* w_scales,
    const uint32_t* w_zeros,
    const int height,
    const int dim,
    const int width,
    const int groupsize,
    const int block_size_z,
    const uint32_t* x_map,
    bool no_zero
)
{
    // Start of block

    int x_column = block_size_z * blockIdx.z;
    int x_column_end = min(dim, block_size_z * (blockIdx.z + 1));

    int w_column = THREADS_X * blockIdx.x + threadIdx.x;
    int x_row = THREADS_Y * blockIdx.y + threadIdx.y;

    int iterations = (x_column_end - x_column) / 8;

    // Views

    MatrixView_half x_(x, height, dim);
    MatrixView_half w_scales_(w_scales, dim / groupsize, width);
    MatrixView_q4_row w_zeros_(w_zeros, dim / groupsize, width);
    MatrixView_q4_column w_(w, dim, width);
    MatrixView_half_rw out_(out, height, width);

    // Zero output

//     if (!no_zero && blockIdx.z == 0)
//     {
//         out_.set(x_row, w_column, {});
//         __syncthreads();
//     }

    if (!no_zero && blockIdx.z == 0 && (threadIdx.x & 1) == 0)
    {
        *((uint32_t*) out_.item_ptr(x_row, w_column)) = 0;
        __syncthreads();
    }

    // Loop over part of x row (and w column)

    half2 acc = {};
    half acc_h = {};

    if constexpr (use_groupsize)
    {
        // For quant matrices where groupsize divides BLOCK_SIZE_Z we always start on a group boundary, so this
        // could be slightly faster

        for (int k = x_column, group = x_column / groupsize; k < x_column + iterations * 8; )
        {
            if constexpr (use_half2)
            {
                half2 w_scale = w_scales_.item_half2half2(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                if constexpr (use_x_map)
                    acc = dot_product_8_x_map(acc, x_, x_row, k, w_, k, w_column, w_scale, w_zero, groupsize / 8, x_map);
                else
                    acc = dot_product_8(acc, x_, x_row, k, w_, k, w_column, w_scale, w_zero, groupsize / 8);

                group++;
                k += groupsize;
            }
            else
            {
                half w_scale = w_scales_.item(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                if constexpr (use_x_map)
                    acc_h = dot_product_8_x_map_h(acc_h, x_, x_row, k, w_, k, w_column, w_scale, w_zero, groupsize / 8, x_map);
                else
                    acc_h = dot_product_8_h(acc_h, x_, x_row, k, w_, k, w_column, w_scale, w_zero, groupsize / 8);

                group++;
                k += groupsize;
            }
        }
    }
    else
    {
        // Otherwise assume groupsize is a multiple of 8, do 8 columns per iteration and trust the cache

        if constexpr (use_half2)
        {
            for (int k = x_column; k < x_column + iterations * 8; )
            {
                int group = k / groupsize;
                half2 w_scale = w_scales_.item_half2half2(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                if constexpr (use_x_map)
                    acc = dot_product_8_x_map(acc, x_, x_row, k, w_, k, w_column, w_scale, w_zero, 1, x_map);
                else
                    acc = dot_product_8(acc, x_, x_row, k, w_, k, w_column, w_scale, w_zero, 1);

                k += 8;
            }
        }
        else
        {
            for (int k = x_column; k < x_column + iterations * 8; )
            {
                int group = k / groupsize;
                half w_scale = w_scales_.item(group, w_column);
                uint32_t w_zero = w_zeros_.item(group, w_column) + 1;

                if constexpr (use_x_map)
                    acc_h = dot_product_8_x_map_h(acc_h, x_, x_row, k, w_, k, w_column, w_scale, w_zero, 1, x_map);
                else
                    acc_h = dot_product_8_h(acc_h, x_, x_row, k, w_, k, w_column, w_scale, w_zero, 1);

                k += 8;
            }
        }
    }

    // Add to block result

    if constexpr (use_half2)
    {
        half result = __hadd(acc.x, acc.y);
        atomicAdd(out_.item_ptr(x_row, w_column), result);
    }
    else
    {
        atomicAdd(out_.item_ptr(x_row, w_column), acc_h);
    }
}

// Compute y = x @ w

void q4_matmul_cuda
(
    ExLlamaTuning* tuningParams,
    const half* x,
    const int x_height,
    const Q4Matrix* w,
    half* out,
    bool no_zero
)
{
    int height = x_height;
    int dim = w->height;
    int width = w->width;

    hipSetDevice(w->device);

    uint32_t* x_map = w->cuda_x_map;
    const half* x_mapped = x;
    if (x_map && !tuningParams->matmul_fused_remap)
    {
        CudaBuffers* buffers = get_buffers(w->device);
        column_remap_cuda(x, buffers->temp_state, x_height, dim, w->cuda_x_map);
        x_mapped = buffers->temp_state;
        x_map = NULL;
    }

    int block_size_z;
    if (w->width == 4096) block_size_z = 384;           // 7B
    else if (w->width == 11008) block_size_z = 256;
    else if (w->width == 5120) block_size_z = 384;      // 13B
    else if (w->width == 13824) block_size_z = 256;
    else if (w->width == 6656) block_size_z = 256;      // 33B
    else if (w->width == 17920) block_size_z = 128;
    else block_size_z = 256;

    //hipMemsetAsync(out, 0, x_height * w->width * sizeof(half));

    dim3 threads(THREADS_X, THREADS_Y, 1);

    dim3 blocks
    (
        (width + threads.x - 1) / threads.x,
        (height + threads.y - 1) / threads.y,
        (dim + block_size_z - 1) / block_size_z
    );

    if (tuningParams->matmul_no_half2)
    {
        if (block_size_z % w->groupsize == 0)
        {
            if (x_map) q4_matmul_kernel <false, true, true>  <<<blocks, threads>>>(x_mapped, w->cuda_qweight, out, w->cuda_scales, w->cuda_qzeros, height, dim, width, w->groupsize, block_size_z, x_map, no_zero);
            else       q4_matmul_kernel <false, true, false> <<<blocks, threads>>>(x_mapped, w->cuda_qweight, out, w->cuda_scales, w->cuda_qzeros, height, dim, width, w->groupsize, block_size_z, NULL, no_zero);
        }
        else
        {
            if (x_map) q4_matmul_kernel <false, false, true>  <<<blocks, threads>>>(x_mapped, w->cuda_qweight, out, w->cuda_scales, w->cuda_qzeros, height, dim, width, w->groupsize, block_size_z, x_map, no_zero);
            else       q4_matmul_kernel <false, false, false> <<<blocks, threads>>>(x_mapped, w->cuda_qweight, out, w->cuda_scales, w->cuda_qzeros, height, dim, width, w->groupsize, block_size_z, NULL, no_zero);
        }
    }
    else
    {
        if (block_size_z % w->groupsize == 0)
        {
            if (x_map) q4_matmul_kernel <true,  true, true>  <<<blocks, threads>>>(x_mapped, w->cuda_qweight, out, w->cuda_scales, w->cuda_qzeros, height, dim, width, w->groupsize, block_size_z, x_map, no_zero);
            else       q4_matmul_kernel <true,  true, false> <<<blocks, threads>>>(x_mapped, w->cuda_qweight, out, w->cuda_scales, w->cuda_qzeros, height, dim, width, w->groupsize, block_size_z, NULL, no_zero);
        }
        else
        {
            if (x_map) q4_matmul_kernel <true,  false, true>  <<<blocks, threads>>>(x_mapped, w->cuda_qweight, out, w->cuda_scales, w->cuda_qzeros, height, dim, width, w->groupsize, block_size_z, x_map, no_zero);
            else       q4_matmul_kernel <true,  false, false> <<<blocks, threads>>>(x_mapped, w->cuda_qweight, out, w->cuda_scales, w->cuda_qzeros, height, dim, width, w->groupsize, block_size_z, NULL, no_zero);
        }
    }
}

void q4_matmul_recons_cuda
(
    ExLlamaTuning* tuningParams,
    const half* x,
    const int x_height,
    Q4Matrix* w,
    half* out,
    const hipblasHandle_t handle
)
{
    int height = x_height;
    int dim = w->height;
    int width = w->width;

    hipSetDevice(w->device);
    CudaBuffers* buffers = get_buffers(w->device);

    const half* x_mapped = x;
    if (w->cuda_x_map)
    {
        column_remap_cuda(x, buffers->temp_state, x_height, dim, w->cuda_x_map);
        x_mapped = buffers->temp_state;
    }

    w->reconstruct(buffers->temp_dq);

    const half alpha = __float2half(1.0f);
    const half beta = __float2half(0.0f);

    hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, width, height, dim, &alpha, buffers->temp_dq, width, x_mapped, dim, &beta, out, width);
}
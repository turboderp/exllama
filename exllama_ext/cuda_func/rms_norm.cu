#include "hip/hip_runtime.h"
#include "rms_norm.cuh"
#include "../cuda_buffers.cuh"
#include "../util.cuh"
#include "../matrix.cuh"

const int THREADS_X = 32;
const int THREADS_Y = 8;
const int BLOCKSIZE_X = 16;

// scratch = sum(x * x, dim = -1)

typedef void (*fp_rms_norm_row_product_kernel)
(
    half*,
    float*,
    const int,
    const int
);

template<bool use_half2>
__global__ void rms_norm_row_product_kernel
(
    half* __restrict__ x,
    float* __restrict__ scratch,
    const int rows,
    const int dim
)
{
    int column = (THREADS_X * blockIdx.x + threadIdx.x) * BLOCKSIZE_X;
    int row = THREADS_Y * blockIdx.y + threadIdx.y;
    if (row >= rows) return;
    if (column >= dim) return;

//     if (column == 0)
//     {
//         scratch[row] = 0.0f;
//         __syncthreads();
//     }

    float acc = 0.0f;
    int idx = row * dim + column;

    // Accumulate

    if constexpr (use_half2)
    {
        half2* x_ptr = (half2*) &x[idx];

        #pragma unroll
        for (int k = 0; k < BLOCKSIZE_X / 2; k++)
        {
            half2 x2 = *x_ptr++;
            float m0 = __low2float(x2);
            float m1 = __high2float(x2);
            acc = fma(m0, m0, acc);
            acc = fma(m1, m1, acc);
        }
    }
    else
    {
        half* x_ptr = x + idx;

        #pragma unroll
        for (int k = 0; k < BLOCKSIZE_X; k++)
        {
            float m0 = __half2float(*x_ptr++);
            acc = fma(m0, m0, acc);
        }
    }

//     // Use Warp Shuffle to accumulate within the warp
//
//     for (int offset = warpSize / 2; offset > 0; offset /= 2)
//         acc += __shfl_down_sync(0xffffffff, acc, offset);
//     if (threadIdx.x % warpSize == 0)
//         atomicAdd(&scratch[row], acc);

    atomicAdd(&scratch[row], acc);
}

// x = x * w / sqrt(scratch / dim + epsilon)

typedef void (*fp_rms_norm_kernel)
(
    half*,
    const half*,
    half*,
    float*,
    const float,
    const float,
    const int,
    const int
);

template<bool use_half2>
__global__ void rms_norm_kernel
(
    half* __restrict__ x,
    const half* __restrict__ w,
    half* __restrict__ out,
    float* __restrict__ scratch,
    const float epsilon,
    const float r_dim,
    const int rows,
    const int dim
)
{
    int column = (THREADS_X * blockIdx.x + threadIdx.x) * BLOCKSIZE_X;
    int row = THREADS_Y * blockIdx.y + threadIdx.y;
    if (row >= rows) return;
    if (column >= dim) return;

    float rmf = rsqrtf(scratch[row] * r_dim + epsilon);
    half rm = __float2half_rn(rmf);
    half2 rm2 = __half2half2(rm);

    if constexpr (use_half2)
    {
        half2* x2_ptr = (half2*) &x[row * dim + column];
        half2* out2_ptr = (half2*) &out[row * dim + column];
        const half2* w2_ptr = (const half2*) &w[column];

        #pragma unroll
        for (int k = 0; k < BLOCKSIZE_X / 2; k++)
        {
            half2 m2 = *x2_ptr++;
            half2 w2 = *w2_ptr++;
            m2 = __hmul2(m2, rm2);
            m2 = __hmul2(m2, w2);
            *out2_ptr++ = m2;
        }
    }
    else
    {
        half* x_ptr = &x[row * dim + column];
        half* out_ptr = &out[row * dim + column];
        const half* w_ptr = &w[column];

        #pragma unroll
        for (int k = 0; k < BLOCKSIZE_X; k++)
        {
            half m = *x_ptr++;
            half w = *w_ptr++;
            m = __hmul(m, rm);
            m = __hmul(m, w);
            *out_ptr++ = m;
        }
    }

//     __syncthreads();
//     if (column >= dim - BLOCKSIZE_X) scratch[row] = 0.0f;
}

fp_rms_norm_row_product_kernel rms_norm_row_product_kernel_pick(ExLlamaTuning* tuningParams)
{
    // <bool use_half2>
    if (tuningParams->matmul_no_half2) {
        return rms_norm_row_product_kernel<false>;
    } else {
        return rms_norm_row_product_kernel<true>;
    }
};

fp_rms_norm_kernel rms_norm_kernel_pick(ExLlamaTuning* tuningParams)
{
    // <bool use_half2>
    if (tuningParams->matmul_no_half2) {
        return rms_norm_kernel<false>;
    } else {
        return rms_norm_kernel<true>;
    }
};

// x = x * w / sqrt(row_mean(x * x) + epsilon)
//
// works in-place if x == out

void rms_norm_cuda
(
    ExLlamaTuning* tuningParams,
    half* x,
    const half* w,
    half* out,
    const float epsilon,
    const int rows,
    const int dim,
    const int device_index
)
{
    CudaBuffers* buffers = get_buffers(device_index);
    float* temp = buffers->get_zeros_float(rows);

    float r_dim = 1.0f / (float) dim;

    dim3 threads(THREADS_X, THREADS_Y, 1);

    dim3 blocks
    (
        ((dim + THREADS_X - 1) / THREADS_X + THREADS_X - 1) / BLOCKSIZE_X,
        (rows + THREADS_Y - 1) / THREADS_Y,
        1
    );

    //hipMemsetAsync(temp, 0, rows * sizeof(float));

    fp_rms_norm_row_product_kernel kernel1 = rms_norm_row_product_kernel_pick(tuningParams);
    kernel1<<<blocks, threads>>>(x, temp, rows, dim);

    fp_rms_norm_kernel kernel2 = rms_norm_kernel_pick(tuningParams);
    kernel2<<<blocks, threads>>>(x, w, out, temp, epsilon, r_dim, rows, dim);

    //hipMemsetAsync(temp, 0, rows * sizeof(float));
}

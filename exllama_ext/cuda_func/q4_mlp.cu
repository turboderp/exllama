#include "hip/hip_runtime.h"
#include "q4_mlp.cuh"
#include "q4_matmul.cuh"
#include "half_matmul.cuh"
#include "rms_norm.cuh"
#include "../cuda_buffers.cuh"
#include "../util.cuh"
#include "../matrix.cuh"
#if defined(USE_ROCM)
#include "../hip_compat.cuh"
#endif

const int THREADS_X = 32;
const int THREADS_Y = 4;
// const int MAX_DIMENSION = 8192;

__device__ __forceinline__ half silu(half x)
{
    half one = __float2half(1.0f);
    half neg_x = __hneg(x);
    half e = hexp(neg_x);
    half sum = __hadd(one, e);
    half r = hrcp(sum);
    half result = __hmul(x, r);
    return result;
}

__device__ __forceinline__ half2 silu(half2 x)
{
    half2 one = __float2half2_rn(1.0f);
    half2 neg_x = __hneg2(x);
    half2 e = h2exp(neg_x);
    half2 sum = __hadd2(one, e);
    half2 r = h2rcp(sum);
    half2 result = __hmul2(x, r);
    return result;
}

typedef void (*fp_silu_mul_cuda_kernel)
(
    half*,
    const half*,
    const int,
    const int
);

template <bool use_half2>
__global__ void silu_mul_cuda_kernel
(
    half* __restrict__ x,
    const half* __restrict__ y,
    const int height,
    const int width
)
{
    MatrixView_half_rw x_(x, height, width);
    MatrixView_half y_(y, height, width);

    int column = (THREADS_X * blockIdx.x + threadIdx.x); if constexpr (use_half2) column *= 2;
    int row = THREADS_Y * blockIdx.y + threadIdx.y;
    if (row >= height) return;

    // silu(x) * y

    if constexpr (use_half2)
    {
        half2 one = __half2half2(__float2half(1.0f));

        half2 x_item = x_.item_half2(row, column);
        half2 y_item = y_.item_half2(row, column);

        x_item = silu(x_item);
        x_item = __hmul2(x_item, y_item);

        x_.set_half2(row, column, x_item);
    }
    else
    {
        half one = __float2half(1.0f);

        half x_item = x_.item(row, column);
        half y_item = y_.item(row, column);

        x_item = silu(x_item);
        x_item = __hmul(x_item, y_item);

        x_.set(row, column, x_item);
    }
}

fp_silu_mul_cuda_kernel silu_mul_cuda_kernel_pick(ExLlamaTuning* tuningParams)
{
    // <bool use_half2>
    if (tuningParams->matmul_no_half2) {
        return silu_mul_cuda_kernel<false>;
    } else {
        return silu_mul_cuda_kernel<true>;
    }
};

void q4_mlp_cuda
(
    ExLlamaTuning* tuningParams,
    half* x,                        // shape == (height, dim)
    const half* rms_norm_weight,    // shape == (x.shape[1],) == (dim,)
    float epsilon,
    Q4Matrix* gate,
    Q4Matrix* up,
    Q4Matrix* down,
    const int height,
    const int dim,
    const half* gate_a,
    const half* gate_b,
    const int gate_rank,
    const half* up_a,
    const half* up_b,
    const int up_rank,
    const half* down_a,
    const half* down_b,
    const int down_rank,
    half* lora_temp,
    hipblasHandle_t handle,
    const int device_index
)
{
    CudaBuffers* buffers = get_buffers(device_index);

    // temp_x = rms_layernorm(x)

    half* temp_x = buffers->temp_state + height * dim;  // TOOD: ..
    TORCH_CHECK(buffers->temp_state_size >= 2 * height * dim, "temp_state buffer too small");
    rms_norm_cuda(tuningParams, x, rms_norm_weight, temp_x, epsilon, height, dim, device_index);

    // temp_mlp[0] = temp_x @ gate
    // temp_mlp[1] = temp_x @ up

    half* temp_mlp_0 = buffers->temp_mlp;
    half* temp_mlp_1 = buffers->temp_mlp + height * up->width;
    int temp_mlp_width = up->width;

    if (gate_a)
    {
        half_matmul_cublas_cuda(tuningParams, temp_x, gate_a, lora_temp, height, dim, gate_rank, handle);
        half_matmul_cublas_cuda(tuningParams, lora_temp, gate_b, temp_mlp_0, height, gate_rank, temp_mlp_width, handle);
    }
    if (up_a)
    {
        half_matmul_cublas_cuda(tuningParams, temp_x, up_a, lora_temp, height, dim, up_rank, handle);
        half_matmul_cublas_cuda(tuningParams, lora_temp, up_b, temp_mlp_1, height, up_rank, temp_mlp_width, handle);
    }

    if (!tuningParams->concurrent_streams)
    {
        q4_matmul_cuda(tuningParams, temp_x, height, gate, temp_mlp_0, gate_a ? true : false);
        q4_matmul_cuda(tuningParams, temp_x, height, up, temp_mlp_1, up_a ? true : false);
    }
    else
    {
        hipStream_t str_1 = buffers->alt_stream_1;
        hipStream_t str_2 = buffers->alt_stream_2;
        hipEvent_t sync_1 = buffers->alt_stream_1_done;
        hipEvent_t sync_2 = buffers->alt_stream_2_done;

        q4_matmul_cuda(tuningParams, temp_x, height, gate, buffers->temp_mlp, gate_a ? true : false, str_1);
        hipEventRecord(sync_1, str_1);

        q4_matmul_cuda(tuningParams, temp_x, height, up, buffers->temp_mlp + height * up->width, up_a ? true : false, str_2);
        hipEventRecord(sync_2, str_2);

        hipStreamWaitEvent(NULL, sync_1, 0);
        hipStreamWaitEvent(NULL, sync_2, 0);
    }

    // temp_mlp[0] = silu(temp_mlp[0]) * temp_mlp[1]

    dim3 threads(THREADS_X, THREADS_Y, 1);

    dim3 blocks
    (
        (up->width + THREADS_X - 1) / THREADS_X / (tuningParams->silu_no_half2 ? 1 : 2),
        (height + THREADS_Y - 1) / THREADS_Y,
        1
    );

    fp_silu_mul_cuda_kernel kernel = silu_mul_cuda_kernel_pick(tuningParams);
    kernel<<<blocks, threads>>>(temp_mlp_0, temp_mlp_1, height, temp_mlp_width);

    // x += temp1 @ down (implicitly add the residual connection by not zeroing the output in the matmul)

    if (down_a)
    {
        half_matmul_cublas_cuda(tuningParams, temp_mlp_0, down_a, lora_temp, height, temp_mlp_width, down_rank, handle);
        half_matmul_cublas_cuda(tuningParams, lora_temp, down_b, x, height, down_rank, dim, handle, true);
    }
    q4_matmul_cuda(tuningParams, temp_mlp_0, height, down, x, true);

    // Reset the temp buffer after use so it's always zeros.
    //hipMemsetAsync(buffers->temp_mlp, 0, 2 * height * up->width * sizeof(half));

}
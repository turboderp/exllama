#include "hip/hip_runtime.h"
#include "q4_mlp.cuh"
#include "q4_matmul.cuh"
#include "rms_norm.cuh"
#include "../cuda_buffers.cuh"
#include "../util.cuh"
#include "../matrix.cuh"
#if defined(USE_ROCM)
#include "../hip_compat.cuh"
#endif

const int THREADS_X = 32;
const int THREADS_Y = 4;
// const int MAX_DIMENSION = 8192;

__device__ __forceinline__ half silu(half x)
{
    half one = __float2half(1.0f);
    half neg_x = __hneg(x);
    half e = hexp(neg_x);
    half sum = __hadd(one, e);
    half r = hrcp(sum);
    half result = __hmul(x, r);
    return result;
}

__device__ __forceinline__ half2 silu(half2 x)
{
    half2 one = __float2half2_rn(1.0f);
    half2 neg_x = __hneg2(x);
    half2 e = h2exp(neg_x);
    half2 sum = __hadd2(one, e);
    half2 r = h2rcp(sum);
    half2 result = __hmul2(x, r);
    return result;
}

template <bool use_half2>
__global__ void silu_mul_cuda_kernel
(
    half* x,
    const half* y,
    const int height,
    const int width
)
{
    MatrixView_half_rw x_(x, height, width);
    MatrixView_half y_(y, height, width);

    int column = (THREADS_X * blockIdx.x + threadIdx.x); if constexpr (use_half2) column *= 2;
    int row = THREADS_Y * blockIdx.y + threadIdx.y;
    if (row >= height) return;

    // silu(x) * y

    if constexpr (use_half2)
    {
        half2 one = __half2half2(__float2half(1.0f));

        half2 x_item = x_.item_half2(row, column);
        half2 y_item = y_.item_half2(row, column);

        x_item = silu(x_item);
        x_item = __hmul2(x_item, y_item);

        x_.set_half2(row, column, x_item);
    }
    else
    {
        half one = __float2half(1.0f);

        half x_item = x_.item(row, column);
        half y_item = y_.item(row, column);

        x_item = silu(x_item);
        x_item = __hmul(x_item, y_item);

        x_.set(row, column, x_item);
    }
}

void q4_mlp_cuda
(
    ExLlamaTuning* tuningParams,
    half* x,                        // shape == (height, dim)
    const half* rms_norm_weight,    // shape == (x.shape[1],) == (dim,)
    float epsilon,
    Q4Matrix* gate,
    Q4Matrix* up,
    Q4Matrix* down,
    const int height,
    const int dim,
    const int device_index
)
{
    CudaBuffers* buffers = get_buffers(device_index);

    // temp_x = rms_layernorm(x)

    half* temp_x = buffers->temp_state + height * dim;
    rms_norm_cuda(tuningParams, x, rms_norm_weight, temp_x, epsilon, height, dim, device_index);

    // temp_mlp[0] = temp_x @ gate
    // temp_mlp[1] = temp_x @ up

    q4_matmul_cuda(tuningParams, temp_x, height, gate, buffers->temp_mlp);
    q4_matmul_cuda(tuningParams, temp_x, height, up, buffers->temp_mlp + height * up->width);

    // temp_mlp[0] = silu(temp_mlp[0]) * temp_mlp[1]

    dim3 threads(THREADS_X, THREADS_Y, 1);

    dim3 blocks
    (
        (up->width + THREADS_X - 1) / THREADS_X / (tuningParams->silu_no_half2 ? 1 : 2),
        (height + THREADS_Y - 1) / THREADS_Y,
        1
    );

    if (tuningParams->silu_no_half2)
    {
        silu_mul_cuda_kernel<false><<<blocks, threads>>>
        (
            buffers->temp_mlp,
            buffers->temp_mlp + height * up->width,
            height,
            up->width
        );
    }
    else
    {
        silu_mul_cuda_kernel<true><<<blocks, threads>>>
        (
            buffers->temp_mlp,
            buffers->temp_mlp + height * up->width,
            height,
            up->width
        );
    }

    // x += temp1 @ down (implicitly add the residual connection by not zeroing the output in the matmul)

    q4_matmul_cuda(tuningParams, buffers->temp_mlp, height, down, x, true);

    // Reset the temp buffer after use so it's always zeros.
    //hipMemsetAsync(buffers->temp_mlp, 0, 2 * height * up->width * sizeof(half));

}
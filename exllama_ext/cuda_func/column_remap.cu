#include "hip/hip_runtime.h"
#include "column_remap.h"
#include "../util.h"

const int SHUF_BLOCKSIZE_X = 256;
const int SHUF_BLOCKSIZE_Y = 16;

__global__ void column_remap_kernel
(
    const half* x,
    half* x_new,
    const int x_width,
    const int x_height,
    const uint32_t* x_map
)
{
    int x_column = SHUF_BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int x_row = SHUF_BLOCKSIZE_Y * blockIdx.y;

    int x_stride = x_width;
    int x_idx = x_row * x_stride + x_column;

    int x_row_end = min(x_row + SHUF_BLOCKSIZE_Y, x_height);
    int x_idx_end = x_row_end * x_stride + x_column;

    int s_column = x_map[x_column];
    int s_idx = x_row * x_stride + s_column;

    while (x_idx < x_idx_end)
    {
        x_new[x_idx] = x[s_idx];
        x_idx += x_stride;
        s_idx += x_stride;
    }
}


// Remap columns in x to correspond to sequential group index before matmul
//
// perform x -> seq_x such that seq_x @ seq_w == x @ w

hipError_t column_remap_cuda
(
    const half* x,
    half* x_new,
    const int x_height,
    const int x_width,
    const uint32_t* x_map
)
{
    hipError_t _cuda_err = hipSuccess;

    dim3 threads(SHUF_BLOCKSIZE_X, 1, 1);

    dim3 blocks
    (
        (x_width + SHUF_BLOCKSIZE_X - 1) / SHUF_BLOCKSIZE_X,
        (x_height + SHUF_BLOCKSIZE_Y - 1) / SHUF_BLOCKSIZE_Y,
        1
    );

    column_remap_kernel<<<blocks, threads>>>(x, x_new, x_width, x_height, x_map);

//     hipDeviceSynchronize();
//     _cuda_check(hipGetLastError());
//
// _cuda_fail:

    return _cuda_err;
}

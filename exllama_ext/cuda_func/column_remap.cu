#include "hip/hip_runtime.h"
#include "column_remap.cuh"
#include "../util.cuh"

// Using 1024 make me crash with "Memory access fault by GPU node-1 (Agent
// handle: 0x012345678912) on address 0x012345678912. Reason: Page not present
// or supervisor privilege."
#if defined(USE_ROCM)
const int SHUF_BLOCKSIZE_X = 256;
#else
const int SHUF_BLOCKSIZE_X = 1024;
#endif
const int SHUF_BLOCKSIZE_Y = 16;

__global__ void column_remap_kernel
(
    const half* x,
    half* x_new,
    const int x_width,
    const int x_height,
    const uint32_t* x_map
)
{
    int x_column = SHUF_BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int x_row = SHUF_BLOCKSIZE_Y * blockIdx.y;

    int x_stride = x_width;
    int x_idx = x_row * x_stride + x_column;

    int x_row_end = min(x_row + SHUF_BLOCKSIZE_Y, x_height);
    int x_idx_end = x_row_end * x_stride + x_column;

    int s_column = x_map[x_column];
    int s_idx = x_row * x_stride + s_column;

    while (x_idx < x_idx_end)
    {
        x_new[x_idx] = x[s_idx];
        x_idx += x_stride;
        s_idx += x_stride;
    }
}


// Remap columns in x to correspond to sequential group index before matmul
//
// perform x -> seq_x such that seq_x @ seq_w == x @ w

void column_remap_cuda
(
    const half* x,
    half* x_new,
    const int x_height,
    const int x_width,
    const uint32_t* x_map
)
{
    dim3 threads(SHUF_BLOCKSIZE_X, 1, 1);

    dim3 blocks
    (
        (x_width + SHUF_BLOCKSIZE_X - 1) / SHUF_BLOCKSIZE_X,
        (x_height + SHUF_BLOCKSIZE_Y - 1) / SHUF_BLOCKSIZE_Y,
        1
    );

    column_remap_kernel<<<blocks, threads>>>(x, x_new, x_width, x_height, x_map);
}
